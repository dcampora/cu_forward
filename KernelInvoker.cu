#include "KernelInvoker.cuh"

extern int*   h_no_sensors;
extern int*   h_no_hits;
extern int*   h_sensor_Zs;
extern int*   h_sensor_hitStarts;
extern int*   h_sensor_hitNums;
extern unsigned int* h_hit_IDs;
extern float* h_hit_Xs;
extern float* h_hit_Ys;
extern float* h_hit_Zs;

hipError_t invokeParallelSearch(
    const int startingEvent,
    const int eventsToProcess,
    const std::vector<const std::vector<uint8_t>* > & input,
    std::vector<std::vector<uint8_t> > & output) {

  // DEBUG << "Input pointer: " 
  //   << std::hex << "0x" << (long long int) &(input[0])
  //   << std::dec << std::endl;
  
  const std::vector<uint8_t>* startingEvent_input = input[startingEvent];
  setHPointersFromInput((uint8_t*) &(*startingEvent_input)[0], startingEvent_input->size());
  // printInfo();

  std::map<int, int> zhit_to_module;
  if (logger::ll.verbosityLevel > 0){
    // map to convert from z of hit to module
    for(int i=0; i<*h_no_sensors; ++i){
      const int z = h_sensor_Zs[i];
      zhit_to_module[z] = i;
    }

    // Some hits z may not correspond to a sensor's,
    // but be close enough
    for(int i=0; i<*h_no_hits; ++i){
      const int z = h_hit_Zs[i];
      if (zhit_to_module.find(z) == zhit_to_module.end()){
        const int sensor = findClosestModule(z, zhit_to_module);
        zhit_to_module[z] = sensor;
      }
    }
  }

  // int* h_prevs, *h_nexts;
  // Histo histo;
  Track* dev_tracks = 0;
  char*  dev_input = 0;
  int*   dev_tracks_to_follow_q1 = 0;
  int*   dev_tracks_to_follow_q2 = 0;
  bool*  dev_hit_used = 0;
  int*   dev_atomicsStorage = 0;
  Track* dev_tracklets = 0;
  int*   dev_weak_tracks = 0;
  int*   dev_event_offsets = 0;
  int*   dev_hit_offsets = 0;

  // Choose which GPU to run on, change this on a multi-GPU system.
  const int module_sides = 2;
  cudaCheck( hipSetDevice(0) );
  dim3 numBlocks(eventsToProcess, module_sides), numThreads(64);

  // Allocate memory
  // Allocate CPU buffers
  int* atomics = (int*) malloc(eventsToProcess * 11 * sizeof(int));

  // Prepare event offset and hit offset
  std::vector<int> event_offsets;
  std::vector<int> hit_offsets;
  int acc_size = 0, acc_hits = 0;
  for (int i=0; i<eventsToProcess; ++i){
    EventBeginning* event = (EventBeginning*) &(*(input[startingEvent + i]))[0];
    const int event_size = input[startingEvent + i]->size();

    event_offsets.push_back(acc_size);
    hit_offsets.push_back(acc_hits);

    acc_size += event_size;
    acc_hits += event->numberOfHits;
  }

  // Allocate GPU buffers
  cudaCheck(hipMalloc((void**)&dev_tracks, eventsToProcess * MAX_TRACKS * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_tracklets, eventsToProcess * module_sides * MAX_TRACKS * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_weak_tracks, eventsToProcess * module_sides * MAX_TRACKS * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_tracks_to_follow_q1, eventsToProcess * module_sides * MAX_TRACKS * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_tracks_to_follow_q2, eventsToProcess * module_sides * MAX_TRACKS * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_atomicsStorage, eventsToProcess * 11 * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_event_offsets, event_offsets.size() * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_hit_offsets, hit_offsets.size() * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_hit_used, acc_hits * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_input, acc_size));

  // Copy stuff from host memory to GPU buffers
  cudaCheck(hipMemcpy(dev_event_offsets, &event_offsets[0], event_offsets.size() * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_hit_offsets, &hit_offsets[0], hit_offsets.size() * sizeof(int), hipMemcpyHostToDevice));

  acc_size = 0;
  for (int i=0; i<eventsToProcess; ++i){
    cudaCheck(hipMemcpy(&dev_input[acc_size], &(*(input[startingEvent + i]))[0], input[startingEvent + i]->size(), hipMemcpyHostToDevice));
    acc_size += input[startingEvent + i]->size();
  }

  // Initialize what we need
  cudaCheck(hipMemset(dev_hit_used, false, acc_hits * sizeof(bool)));
  cudaCheck(hipMemset(dev_atomicsStorage, 0, eventsToProcess * 11 * sizeof(int)));

  // searchByTriplet
  DEBUG << "Now, on your favourite GPU: searchByTriplet with " << eventsToProcess << " event"
    << (eventsToProcess>1 ? "s" : "") << "..." << std::endl;
  hipEvent_t start_searchByTriplet, stop_searchByTriplet;
  float t0;

  hipEventCreate(&start_searchByTriplet);
  hipEventCreate(&stop_searchByTriplet);

  hipEventRecord(start_searchByTriplet, 0 );
  
  searchByTriplet<<<numBlocks, numThreads>>>(dev_tracks, dev_input, dev_tracks_to_follow_q1, dev_tracks_to_follow_q2,
    dev_hit_used, dev_atomicsStorage, dev_tracklets, dev_weak_tracks, dev_event_offsets, dev_hit_offsets);

  hipEventRecord( stop_searchByTriplet, 0 );
  hipEventSynchronize( stop_searchByTriplet );
  hipEventElapsedTime( &t0, start_searchByTriplet, stop_searchByTriplet );

  hipEventDestroy( start_searchByTriplet );
  hipEventDestroy( stop_searchByTriplet );

  cudaCheck( hipPeekAtLastError() );

  DEBUG << "Done!" << std::endl;

  // Get results
  DEBUG << "Number of tracks found per event:" << std::endl << " ";
  cudaCheck(hipMemcpy(atomics, dev_atomicsStorage, eventsToProcess * 11 * sizeof(int), hipMemcpyDeviceToHost));
  for (int i=0; i<eventsToProcess; ++i){
    const int numberOfTracks = atomics[i];
    DEBUG << numberOfTracks << ", ";
    
    output[startingEvent + i].resize(numberOfTracks * sizeof(Track));
    cudaCheck(hipMemcpy(&(output[startingEvent + i])[0], &dev_tracks[i * MAX_TRACKS], numberOfTracks * sizeof(Track), hipMemcpyDeviceToHost));
  }
  DEBUG << std::endl;

  // Print info about the solution
  // const int numberOfTracks = output[0].size() / sizeof(Track);
  // Track* tracks_in_solution = (Track*) &(output[0])[0];
  // print debug info
  // if (logger::ll.verbosityLevel > 0){
  //   for(int i=0; i<numberOfTracks; ++i){
  //     printTrack(tracks_in_solution, i, zhit_to_module);
  //   }
  // }

  // DEBUG << "Got " << numberOfTracks << " tracks" << std::endl;
  DEBUG << "It took " << t0 << " milliseconds." << std::endl;

  free(atomics);

  // hipError_t cudaStatus = hipSuccess;
  return hipSuccess;
}

/**
 * Prints tracks
 * Track #n, length <length>:
 *  <ID> module <module>, x <x>, y <y>, z <z>
 * 
 * @param tracks      
 * @param trackNumber 
 */
void printTrack(Track* tracks, const int trackNumber, const std::map<int, int>& zhit_to_module){
  const Track t = tracks[trackNumber];
  DEBUG << "Track #" << trackNumber << ", length " << (int) t.hitsNum << std::endl;

  for(int i=0; i<t.hitsNum; ++i){
    const int hitNumber = t.hits[i];
    const unsigned int id = h_hit_IDs[hitNumber];
    const float x = h_hit_Xs[hitNumber];
    const float y = h_hit_Ys[hitNumber];
    const float z = h_hit_Zs[hitNumber];
    const int module = zhit_to_module.at((int) z);

    DEBUG << " " << std::setw(8) << id
      << " module " << std::setw(2) << module
      << ", x " << std::setw(6) << x
      << ", y " << std::setw(6) << y
      << ", z " << std::setw(6) << z << std::endl;
  }

  DEBUG << std::endl;
}

/**
 * The z of the hit may not correspond to any z in the sensors.
 * @param  z              
 * @param  zhit_to_module 
 * @return                sensor number
 */
int findClosestModule(const int z, const std::map<int, int>& zhit_to_module){
  if (zhit_to_module.find(z) != zhit_to_module.end())
    return zhit_to_module.at(z);

  int error = 0;
  while(true){
    error++;
    const int lowerAttempt = z - error;
    const int higherAttempt = z + error;

    if (zhit_to_module.find(lowerAttempt) != zhit_to_module.end()){
      return zhit_to_module.at(lowerAttempt);
    }
    if (zhit_to_module.find(higherAttempt) != zhit_to_module.end()){
      return zhit_to_module.at(higherAttempt);
    }
  }
}

void printOutAllSensorHits(int* prevs, int* nexts){
  DEBUG << "All valid sensor hits: " << std::endl;
  for(int i=0; i<h_no_sensors[0]; ++i){
    for(int j=0; j<h_sensor_hitNums[i]; ++j){
      int hit = h_sensor_hitStarts[i] + j;

      if(nexts[hit] != -1){
        DEBUG << hit << ", " << nexts[hit] << std::endl;
      }
    }
  }
}

void printOutSensorHits(int sensorNumber, int* prevs, int* nexts){
  for(int i=0; i<h_sensor_hitNums[sensorNumber]; ++i){
    int hstart = h_sensor_hitStarts[sensorNumber];

    DEBUG << hstart + i << ": " << prevs[hstart + i] << ", " << nexts[hstart + i] << std::endl;
  }
}

void printInfo() {
  DEBUG << "Read info:" << std::endl
    << " no sensors: " << h_no_sensors[0] << std::endl
    << " no hits: " << h_no_hits[0] << std::endl
    << "First 5 sensors: " << std::endl;

  for (int i=0; i<5; ++i){
    DEBUG << " Zs: " << h_sensor_Zs[i] << std::endl
      << " hitStarts: " << h_sensor_hitStarts[i] << std::endl
      << " hitNums: " << h_sensor_hitNums[i] << std::endl << std::endl;
  }

  DEBUG << "First 5 hits: " << std::endl;

  for (int i=0; i<5; ++i){
    DEBUG << " hit_id: " << h_hit_IDs[i] << std::endl
      << " hit_X: " << h_hit_Xs[i] << std::endl
      << " hit_Y: " << h_hit_Ys[i] << std::endl
      << " hit_Z: " << h_hit_Zs[i] << std::endl << std::endl;
  }
}

void getMaxNumberOfHits(char*& input, int& maxHits){
  int* l_no_sensors = (int*) &input[0];
  int* l_no_hits = (int*) (l_no_sensors + 1);
  int* l_sensor_Zs = (int*) (l_no_hits + 1);
  int* l_sensor_hitStarts = (int*) (l_sensor_Zs + l_no_sensors[0]);
  int* l_sensor_hitNums = (int*) (l_sensor_hitStarts + l_no_sensors[0]);

  maxHits = 0;
  for(int i=0; i<l_no_sensors[0]; ++i){
    if(l_sensor_hitNums[i] > maxHits)
      maxHits = l_sensor_hitNums[i];
  }
}
