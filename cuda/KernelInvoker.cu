#include "hip/hip_runtime.h"
#include "KernelInvoker.cuh"

hipError_t invokeParallelSearch(
  const std::vector<std::vector<uint8_t>>& input,
  std::vector<std::vector<uint8_t>>& output
) {
  int eventsToProcess = input.size();

  // int* h_prevs, *h_nexts;
  // Histo histo;
  Track* dev_tracks;
  char*  dev_input;
  int*   dev_tracks_to_follow;
  bool*  dev_hit_used;
  int*   dev_atomicsStorage;
  Track* dev_tracklets;
  int*   dev_weak_tracks;
  int*   dev_event_offsets;
  int*   dev_hit_offsets;
  float* dev_best_fits;
  int*   dev_hit_candidates;
  int*   dev_hit_h2_candidates;

  // Choose which GPU to run on, change this on a multi-GPU system.
  const int device_number = 0;

  cudaCheck(hipSetDevice(device_number));
  cudaCheck(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

  hipDeviceProp_t* device_properties = (hipDeviceProp_t*) malloc(sizeof(hipDeviceProp_t));
  hipGetDeviceProperties(device_properties, 0);

  // Some startup settings
  dim3 numBlocks(eventsToProcess);
  dim3 numThreads(NUMTHREADS_X, 4);

  // Allocate memory
  // Prepare event offset and hit offset
  std::vector<int> event_offsets;
  std::vector<int> hit_offsets;
  int acc_size = 0, acc_hits = 0;
  for (int i=0; i<eventsToProcess; ++i) {
    auto info = EventInfo(input[i]);
    const int event_size = input[i].size();
    event_offsets.push_back(acc_size);
    hit_offsets.push_back(acc_hits);
    acc_size += event_size;
    acc_hits += info.numberOfHits;
  }

  // Allocate CPU buffers
  const int atomic_space = NUM_ATOMICS + 1;
  int* atomics = (int*) malloc(eventsToProcess * atomic_space * sizeof(int));  
  int* hit_candidates = (int*) malloc(2 * acc_hits * sizeof(int));

  // Allocate GPU buffers
  cudaCheck(hipMalloc((void**)&dev_tracks, eventsToProcess * MAX_TRACKS * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_tracklets, acc_hits * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_weak_tracks, acc_hits * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_tracks_to_follow, eventsToProcess * TTF_MODULO * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_atomicsStorage, eventsToProcess * atomic_space * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_event_offsets, event_offsets.size() * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_hit_offsets, hit_offsets.size() * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_hit_used, acc_hits * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_input, acc_size));
  cudaCheck(hipMalloc((void**)&dev_best_fits, eventsToProcess * numThreads.x * MAX_NUMTHREADS_Y * sizeof(float)));
  cudaCheck(hipMalloc((void**)&dev_hit_candidates, 2 * acc_hits * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_hit_h2_candidates, 2 * acc_hits * sizeof(int)));

  // Copy stuff from host memory to GPU buffers
  cudaCheck(hipMemcpy(dev_event_offsets, &event_offsets[0], event_offsets.size() * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_hit_offsets, &hit_offsets[0], hit_offsets.size() * sizeof(int), hipMemcpyHostToDevice));

  acc_size = 0;
  for (int i=0; i<eventsToProcess; ++i){
    cudaCheck(hipMemcpy(&dev_input[acc_size], input[i].data(), input[i].size(), hipMemcpyHostToDevice));
    acc_size += input[i].size();
  }

  // Adding timing
  // Timing calculation
  unsigned int niterations = 3;
  unsigned int nexperiments = 5;

  std::vector<std::vector<float>> time_values {nexperiments};
  std::vector<std::map<std::string, float>> mresults {nexperiments};

  DEBUG << "Now, on your " << device_properties->name << ": searchByTriplet with " << eventsToProcess << " event" << (eventsToProcess>1 ? "s" : "") << std::endl 
	  << " " << nexperiments << " experiments, " << niterations << " iterations" << std::endl;

  if (nexperiments!=1) {
    numThreads.y = 1;
  }

  for (auto i=0; i<nexperiments; ++i) {

    DEBUG << numThreads.x << ", " << numThreads.y << ": " << std::flush;

    for (auto j=0; j<niterations; ++j) {
      // Initialize what we need
      cudaCheck(hipMemset(dev_hit_used, false, acc_hits * sizeof(bool)));
      cudaCheck(hipMemset(dev_atomicsStorage, 0, eventsToProcess * atomic_space * sizeof(int)));
      cudaCheck(hipMemset(dev_hit_candidates, -1, 2 * acc_hits * sizeof(int)));
      cudaCheck(hipMemset(dev_hit_h2_candidates, -1, 2 * acc_hits * sizeof(int)));

      // Just for debugging purposes
      cudaCheck(hipMemset(dev_tracks, 0, eventsToProcess * MAX_TRACKS * sizeof(Track)));
      cudaCheck(hipMemset(dev_tracklets, 0, acc_hits * sizeof(Track)));
      cudaCheck(hipMemset(dev_tracks_to_follow, 0, eventsToProcess * TTF_MODULO * sizeof(int)));
      cudaCheck(hipDeviceSynchronize());

      // searchByTriplet
      hipEvent_t start_searchByTriplet, stop_searchByTriplet;
      float t0;

      hipEventCreate(&start_searchByTriplet);
      hipEventCreate(&stop_searchByTriplet);

      hipEventRecord(start_searchByTriplet, 0 );
      
      searchByTriplet<<<numBlocks, numThreads>>>(dev_tracks, (const char*) dev_input,
        dev_tracks_to_follow, dev_hit_used, dev_atomicsStorage, dev_tracklets,
        dev_weak_tracks, dev_event_offsets, dev_hit_offsets, dev_best_fits,
        dev_hit_candidates, dev_hit_h2_candidates);

      hipEventRecord( stop_searchByTriplet, 0 );
      hipEventSynchronize( stop_searchByTriplet );
      hipEventElapsedTime( &t0, start_searchByTriplet, stop_searchByTriplet );

      hipEventDestroy( start_searchByTriplet );
      hipEventDestroy( stop_searchByTriplet );

      cudaCheck( hipPeekAtLastError() );

      time_values[i].push_back(t0);

      DEBUG << "." << std::flush;
    }
    
    if (nexperiments!=1) {
      numThreads.y *= 2;
    }

    DEBUG << std::endl;
  }

  // Get results
  if (PRINT_SOLUTION) DEBUG << "Number of tracks found per event:" << std::endl << " ";
  cudaCheck(hipMemcpy(atomics, dev_atomicsStorage, eventsToProcess * atomic_space * sizeof(int), hipMemcpyDeviceToHost));
  for (int i=0; i<eventsToProcess; ++i){
    const int numberOfTracks = atomics[i];
    if (PRINT_SOLUTION) DEBUG << numberOfTracks << ", ";

    std::vector<uint8_t> output_track (numberOfTracks * sizeof(Track));
    cudaCheck(hipMemcpy(output_track.data(), &dev_tracks[i * MAX_TRACKS], numberOfTracks * sizeof(Track), hipMemcpyDeviceToHost));
    output.push_back(output_track);
  }
  if (PRINT_SOLUTION) DEBUG << std::endl;

  if (PRINT_VERBOSE) {
    // Print solution of all events processed, to results
    for (int i=0; i<eventsToProcess; ++i) {

      // Print to output file with event no.
      const int numberOfTracks = output[i].size() / sizeof(Track);
      Track* tracks_in_solution = (Track*) &(output[i])[0];
      std::ofstream outfile (std::string(RESULTS_FOLDER) + std::string("/") + std::to_string(i) + std::string(".out"));
      for(int j=0; j<numberOfTracks; ++j){
        printTrack(EventInfo(input[i]), tracks_in_solution, j, outfile);
      }
      outfile.close();
    }
  }

  if (PRINT_BINARY) {
    std::cout << "Printing binary solution" << std::endl;
    for (int i=0; i<eventsToProcess; ++i) {
      const int numberOfTracks = output[i].size() / sizeof(Track);
      Track* tracks_in_solution = (Track*) &(output[i])[0];

      std::ofstream outfile (std::string(RESULTS_FOLDER) + std::string("/tracks_") + std::to_string(i) + std::string(".bin"), std::ios::binary);
      outfile.write((char*) &numberOfTracks, sizeof(int32_t));
      for(int j=0; j<numberOfTracks; ++j){
        writeBinaryTrack(EventInfo(input[i]), tracks_in_solution[j], outfile);
      }
      outfile.close();

      if ((i%100) == 0) {
        std::cout << "." << std::flush;
      }
    }
    std::cout << std::endl;
  }

  DEBUG << std::endl << "Time averages:" << std::endl;
  int exp = 1;
  for (auto i=0; i<nexperiments; ++i){
    mresults[i] = calcResults(time_values[i]);
    DEBUG << " nthreads (" << NUMTHREADS_X << ", " << (nexperiments==1 ? numThreads.y : exp) <<  "): "
      << eventsToProcess / (mresults[i]["mean"] * 0.001) << " events/s, "
      << mresults[i]["mean"] << " ms (std dev " << mresults[i]["deviation"] << ")" << std::endl;

    exp *= 2;
  }

  free(atomics);

  return hipSuccess;
}
