#include "hip/hip_runtime.h"
#include "Tools.cuh"

/**
 * @brief Obtains results statistics.
 */
std::map<std::string, float> calcResults(std::vector<float>& times){
    // sqrt ( E( (X - m)2) )
    std::map<std::string, float> results;
    float deviation = 0.0f, variance = 0.0f, mean = 0.0f, min = FLT_MAX, max = 0.0f;

    for(auto it = times.begin(); it != times.end(); it++){
        const float seconds = (*it);
        mean += seconds;
        variance += seconds * seconds;

        if (seconds < min) min = seconds;
        if (seconds > max) max = seconds;
    }

    mean /= times.size();
    variance = (variance / times.size()) - (mean * mean);
    deviation = std::sqrt(variance);

    results["variance"] = variance;
    results["deviation"] = deviation;
    results["mean"] = mean;
    results["min"] = min;
    results["max"] = max;

    return results;
}

/**
 * @brief Writes a track in binary format
 * 
 * @details The binary format is per every track:
 *   hitsNum hit0 hit1 hit2 ... (#hitsNum times)
 */
void writeBinaryTrack(
  const EventInfo& info,
  const Track& track,
  std::ofstream& outstream
) {
  outstream.write((char*) &track.hitsNum, sizeof(uint32_t));
  for (int i=0; i<track.hitsNum; ++i) {
    outstream.write((char*) &info.hit_IDs[track.hits[i]], sizeof(uint32_t));
  }
}

/**
 * Prints tracks
 * Track #n, length <length>:
 *  <ID> module <module>, x <x>, y <y>, z <z>
 * 
 * @param tracks      
 * @param trackNumber 
 */
void printTrack(
  const EventInfo& info,
  Track* tracks,
  const int trackNumber,
  std::ofstream& outstream
) {
  const Track t = tracks[trackNumber];
  outstream << "Track #" << trackNumber << ", length " << (int) t.hitsNum << std::endl;

  for(int i=0; i<t.hitsNum; ++i){
    const int hitNumber = t.hits[i];
    const unsigned int id = info.hit_IDs[hitNumber];
    const float x = info.hit_Xs[hitNumber];
    const float y = info.hit_Ys[hitNumber];

    int module = 0;
    for (int i=0; i<info.numberOfSensors; ++i) {
      if (hitNumber >= info.sensor_hitStarts[i] &&
          hitNumber < info.sensor_hitStarts[i] + info.sensor_hitNums[i]) {
        module = i;
      }
    }

    outstream << " " << std::setw(8) << id << " (" << hitNumber << ")"
      << " module " << std::setw(2) << module
      << ", x " << std::setw(6) << x
      << ", y " << std::setw(6) << y
      << ", z " << std::setw(6) << info.sensor_Zs[module]
      << std::endl;
  }

  outstream << std::endl;
}

void printOutAllSensorHits(const EventInfo& info, int* prevs, int* nexts) {
  DEBUG << "All valid sensor hits: " << std::endl;
  for(int i=0; i<info.numberOfSensors; ++i){
    for(int j=0; j<info.sensor_hitNums[i]; ++j){
      int hit = info.sensor_hitStarts[i] + j;

      if(nexts[hit] != -1){
        DEBUG << hit << ", " << nexts[hit] << std::endl;
      }
    }
  }
}

void printOutSensorHits(const EventInfo& info, int sensorNumber, int* prevs, int* nexts){
  for(int i=0; i<info.sensor_hitNums[sensorNumber]; ++i){
    int hstart = info.sensor_hitStarts[sensorNumber];

    DEBUG << hstart + i << ": " << prevs[hstart + i] << ", " << nexts[hstart + i] << std::endl;
  }
}

void printInfo(const EventInfo& info, int numberOfSensors, int numberOfHits) {
  numberOfSensors = numberOfSensors>52 ? 52 : numberOfSensors;

  DEBUG << "Read info:" << std::endl
    << " no sensors: " << info.numberOfSensors << std::endl
    << " no hits: " << info.numberOfHits << std::endl
    << numberOfSensors << " sensors: " << std::endl;

  for (int i=0; i<numberOfSensors; ++i){
    DEBUG << " Zs: " << info.sensor_Zs[i] << std::endl
      << " hitStarts: " << info.sensor_hitStarts[i] << std::endl
      << " hitNums: " << info.sensor_hitNums[i] << std::endl << std::endl;
  }

  DEBUG << numberOfHits << " hits: " << std::endl;

  for (int i=0; i<numberOfHits; ++i){
    DEBUG << " hit_id: " << info.hit_IDs[i] << std::endl
      << " hit_X: " << info.hit_Xs[i] << std::endl
      << " hit_Y: " << info.hit_Ys[i] << std::endl
      // << " hit_Z: " << info.hit_Zs[i] << std::endl
      << std::endl;
  }
}
