#include "hip/hip_runtime.h"
#include "../include/KernelInvoker.cuh"

hipError_t invokeParallelSearch(
  const std::vector<std::vector<uint8_t>>& input,
  std::vector<std::vector<uint8_t>>& output
) {
  unsigned int eventsToProcess = input.size();

  // Choose which GPU to run on
  const int device_number = 0;
  cudaCheck(hipSetDevice(device_number));
  cudaCheck(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  hipDeviceProp_t* device_properties = (hipDeviceProp_t*) malloc(sizeof(hipDeviceProp_t));
  hipGetDeviceProperties(device_properties, 0);

  // Blocks and threads
  dim3 numBlocks(eventsToProcess);
  dim3 numThreads(NUMTHREADS_X);

  // Allocate memory
  // Prepare event offset and hit offset
  std::vector<unsigned int> event_offsets;
  std::vector<unsigned int> hit_offsets;
  int acc_size = 0, acc_hits = 0;
  for (unsigned int i=0; i<eventsToProcess; ++i) {
    auto info = EventInfo(input[i]);
    const int event_size = input[i].size();
    event_offsets.push_back(acc_size);
    hit_offsets.push_back(acc_hits);
    acc_size += event_size;
    acc_hits += info.numberOfHits;
  }

  // Number of defined atomics
  constexpr unsigned int atomic_space = NUM_ATOMICS + 1;

  // GPU datatypes
  Track* dev_tracks;
  char* dev_input;
  unsigned int* dev_tracks_to_follow;
  bool* dev_hit_used;
  int* dev_atomicsStorage;
  Track* dev_tracklets;
  unsigned int* dev_weak_tracks;
  unsigned int* dev_event_offsets;
  unsigned int* dev_hit_offsets;
  short* dev_h0_candidates;
  short* dev_h2_candidates;
  unsigned short* dev_rel_indices;
  float* dev_hit_phi;
  int32_t* dev_hit_temp;
  unsigned short* dev_hit_permutation;

  // Allocate GPU buffers
  cudaCheck(hipMalloc((void**)&dev_tracks, eventsToProcess * MAX_TRACKS * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_input, acc_size));
  cudaCheck(hipMalloc((void**)&dev_tracks_to_follow, eventsToProcess * TTF_MODULO * sizeof(unsigned int)));
  cudaCheck(hipMalloc((void**)&dev_hit_used, acc_hits * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_atomicsStorage, eventsToProcess * atomic_space * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_tracklets, acc_hits * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_weak_tracks, acc_hits * sizeof(unsigned int)));
  cudaCheck(hipMalloc((void**)&dev_event_offsets, event_offsets.size() * sizeof(unsigned int)));
  cudaCheck(hipMalloc((void**)&dev_hit_offsets, hit_offsets.size() * sizeof(unsigned int)));
  cudaCheck(hipMalloc((void**)&dev_h0_candidates, 2 * acc_hits * sizeof(short)));
  cudaCheck(hipMalloc((void**)&dev_h2_candidates, 2 * acc_hits * sizeof(short)));
  cudaCheck(hipMalloc((void**)&dev_rel_indices, eventsToProcess * MAX_NUMHITS_IN_MODULE * sizeof(unsigned short)));
  cudaCheck(hipMalloc((void**)&dev_hit_phi, acc_hits * sizeof(float)));
  cudaCheck(hipMalloc((void**)&dev_hit_temp, acc_hits * sizeof(int32_t)));
  cudaCheck(hipMalloc((void**)&dev_hit_permutation, acc_hits * sizeof(unsigned short)));

  // Copy stuff from host memory to GPU buffers
  cudaCheck(hipMemcpy(dev_event_offsets, event_offsets.data(), event_offsets.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_hit_offsets, hit_offsets.data(), hit_offsets.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
  acc_size = 0;
  for (unsigned int event_no=0; event_no<eventsToProcess; ++event_no){
    cudaCheck(hipMemcpy(&dev_input[acc_size], input[event_no].data(), input[event_no].size(), hipMemcpyHostToDevice));
    acc_size += input[event_no].size();
  }

  // // Sorting
  // hipEvent_t start_sort, stop_sort;
  // float tsort;
  // hipEventCreate(&start_sort);
  // hipEventCreate(&stop_sort);
  // hipEventRecord(start_sort, 0);

  // calculatePhiAndSort<<<numBlocks, 64>>>(
  //   (const char*) dev_input,
  //   dev_event_offsets,
  //   dev_hit_offsets,
  //   dev_hit_phi,
  //   dev_hit_temp,
  //   dev_hit_permutation
  // );

  // hipEventRecord(stop_sort, 0);
  // hipEventSynchronize(stop_sort);
  // hipEventElapsedTime(&tsort, start_sort, stop_sort);
  // hipEventDestroy(start_sort);
  // hipEventDestroy(stop_sort);

  // Repeat the processing several times to average time
  unsigned int niterations = 1;
  unsigned int nexperiments = 1;
  std::vector<std::vector<float>> time_values {nexperiments};
  std::vector<std::map<std::string, float>> mresults {nexperiments};

  DEBUG << "Now, on your " << device_properties->name
    << ": searchByTriplet with " << eventsToProcess
    << " event" << (eventsToProcess>1 ? "s" : "") << std::endl 
	  << " " << nexperiments << " experiments, "
    << niterations << " iterations" << std::endl;

  for (auto i=0; i<nexperiments; ++i) {

    DEBUG << numThreads.x << ": " << std::flush;

    for (auto j=0; j<niterations; ++j) {
      // Initialize just what we need
      cudaCheck(hipMemset(dev_hit_used, false, acc_hits * sizeof(bool)));
      cudaCheck(hipMemset(dev_atomicsStorage, 0, eventsToProcess * atomic_space * sizeof(int)));
      
      // searchByTriplet
      hipEvent_t start_searchByTriplet, stop_searchByTriplet;
      float t0;

      hipEventCreate(&start_searchByTriplet);
      hipEventCreate(&stop_searchByTriplet);

      hipEventRecord(start_searchByTriplet, 0 );

      searchByTriplet<<<numBlocks, numThreads>>>(
        dev_tracks,
        dev_input,
        dev_tracks_to_follow,
        dev_hit_used,
        dev_atomicsStorage,
        dev_tracklets,
        dev_weak_tracks,
        dev_event_offsets,
        dev_hit_offsets,
        dev_h0_candidates,
        dev_h2_candidates,
        dev_rel_indices,
        dev_hit_phi,
        dev_hit_temp,
        dev_hit_permutation
      );

      hipEventRecord( stop_searchByTriplet, 0 );
      hipEventSynchronize( stop_searchByTriplet );
      hipEventElapsedTime( &t0, start_searchByTriplet, stop_searchByTriplet );

      hipEventDestroy( start_searchByTriplet );
      hipEventDestroy( stop_searchByTriplet );

      cudaCheck( hipPeekAtLastError() );

      time_values[i].push_back(t0);

      DEBUG << "." << std::flush;

    }

    DEBUG << std::endl;
  }

  if (PRINT_FILL_CANDIDATES) {
    std::vector<short> h0_candidates (2 * acc_hits);
    std::vector<short> h2_candidates (2 * acc_hits);
    cudaCheck(hipMemcpy(h0_candidates.data(), dev_h0_candidates, 2 * acc_hits * sizeof(short), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(h2_candidates.data(), dev_h2_candidates, 2 * acc_hits * sizeof(short), hipMemcpyDeviceToHost));
    
    // Just print modules 49, 47 and 45
    auto info = EventInfo(input[0]);

    std::vector<unsigned int> modules {49, 47, 45};
    for (auto module : modules) {
      std::cout << "Module " << module << std::endl << " h0 candidates: ";
      for (auto i=info.module_hitStarts[module]; i<info.module_hitStarts[module]+info.module_hitNums[module]; ++i) {
        std::cout << "(" << h0_candidates[2*i] << ", " << h0_candidates[2*i+1] << ") ";
      }
      std::cout << std::endl;
    }
    
    for (auto module : modules) {
      std::cout << "Module " << module << std::endl << " h2 candidates: ";
      for (auto i=info.module_hitStarts[module]; i<info.module_hitStarts[module]+info.module_hitNums[module]; ++i) {
        std::cout << "(" << h2_candidates[2*i] << ", " << h2_candidates[2*i+1] << ") ";
      }
      std::cout << std::endl;
    }
  }

  // Get results
  if (PRINT_SOLUTION) DEBUG << "Number of tracks found per event:" << std::endl << " ";
  std::vector<int> atomics (eventsToProcess * atomic_space);
  cudaCheck(hipMemcpy(atomics.data(), dev_atomicsStorage, eventsToProcess * atomic_space * sizeof(int), hipMemcpyDeviceToHost));
  for (unsigned int i=0; i<eventsToProcess; ++i){
    const unsigned int numberOfTracks = atomics[i];
    if (PRINT_SOLUTION) DEBUG << numberOfTracks << ", ";

    std::vector<uint8_t> output_track (numberOfTracks * sizeof(Track));
    cudaCheck(hipMemcpy(output_track.data(), &dev_tracks[i * MAX_TRACKS], numberOfTracks * sizeof(Track), hipMemcpyDeviceToHost));
    output.push_back(output_track);
  }
  if (PRINT_SOLUTION) DEBUG << std::endl;

  if (PRINT_VERBOSE) {
    // Print solution of all events processed, to results
    for (unsigned int i=0; i<eventsToProcess; ++i) {

      // Print to output file with event no.
      const int numberOfTracks = output[i].size() / sizeof(Track);
      Track* tracks_in_solution = (Track*) &(output[i])[0];
      std::ofstream outfile (std::string(RESULTS_FOLDER) + std::string("/") + std::to_string(i) + std::string(".txt"));
      for(int j=0; j<numberOfTracks; ++j){
        printTrack(EventInfo(input[i]), tracks_in_solution, j, outfile);
      }
      outfile.close();
    }
  }

  if (PRINT_BINARY) {
    std::cout << "Printing binary solution" << std::endl;
    for (unsigned int i=0; i<eventsToProcess; ++i) {
      const int numberOfTracks = output[i].size() / sizeof(Track);
      Track* tracks_in_solution = (Track*) &(output[i])[0];

      std::ofstream outfile (std::string(RESULTS_FOLDER) + std::string("/") + std::to_string(i) + std::string(".bin"), std::ios::binary);
      outfile.write((char*) &numberOfTracks, sizeof(int32_t));
      // Fetch back the event
      std::vector<uint8_t> event_data (input[i].size());
      cudaCheck(hipMemcpy(event_data.data(), &dev_input[event_offsets[i]], event_data.size(), hipMemcpyDeviceToHost));
      auto info = EventInfo(event_data);
      for(int j=0; j<numberOfTracks; ++j){
        writeBinaryTrack((unsigned int*) info.hit_Zs, tracks_in_solution[j], outfile);
      }
      outfile.close();

      if ((i%100) == 0) {
        std::cout << "." << std::flush;
      }
    }
    std::cout << std::endl;
  }

  // DEBUG << std::endl << "Time averages:" << std::endl
  //   << " Phi + sorting throughput: " << eventsToProcess / (tsort * 0.001)
  //   << " events/s, (" << tsort << " ms)" << std::endl;

  int exp = 1;
  for (auto i=0; i<nexperiments; ++i){
    mresults[i] = calcResults(time_values[i]);
    DEBUG << " nthreads (" << NUMTHREADS_X << "): "
      << eventsToProcess / (mresults[i]["mean"] * 0.001) << " events/s, "
      << mresults[i]["mean"] << " ms (std dev " << mresults[i]["deviation"] << "), "
      // << eventsToProcess / ((mresults[i]["mean"] + tsort) * 0.001) << " events/s with sorting"
      << std::endl;

    exp *= 2;
  }

  return hipSuccess;
}
