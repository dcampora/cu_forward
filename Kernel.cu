#include "hip/hip_runtime.h"
﻿
#include "Kernel.cuh"

/**
 * @brief Gives the fit between h0, h1 and h2
 * @details The result is given in a float. MAX_FLOAT is
 *          used as an upper limit. The lower it is, the better the fit is.
 *          
 * @param  h0 
 * @param  h1 
 * @param  h2 
 * @return    
 */
__device__ float fitHits(const Hit& h0, const Hit& h1, const Hit &h2, const float dxmax, const float dymax) {
  // Max dx, dy permissible over next hit

  // First approximation -
  // With the sensor z, instead of the hit z
  const float z2_tz = (h2.z - h0.z) / (h1.z - h0.z);
  const float x = h0.x + (h1.x - h0.x) * z2_tz;
  const float y = h0.y + (h1.y - h0.y) * z2_tz;

  const float dx = x - h2.x;
  const float dy = y - h2.y;

  // Scatter - Updated to last PrPixel
  const float scatterNum = (dx * dx) + (dy * dy);
  const float scatterDenom = 1.f / (h2.z - h1.z);
  const float scatter = scatterNum * scatterDenom * scatterDenom;

  const bool scatter_condition = scatter < MAX_SCATTER;
  const bool condition = fabs(h1.x - h0.x) < dxmax && fabs(h1.y - h0.y) < dymax && scatter_condition;

  return condition * scatter + !condition * MAX_FLOAT;
}

/**
 * @brief Fits hits to tracks.
 * @details In case the tolerances constraints are met,
 *          returns the chi2 weight of the track. Otherwise,
 *          returns MAX_FLOAT.
 * 
 * @param tx 
 * @param ty 
 * @param h0 
 * @param h1_z
 * @param h2 
 * @return 
 */
__device__ float fitHitToTrack(const float tx, const float ty, const Hit& h0, const float h1_z, const Hit& h2){
  // tolerances
  const float dz = h2.z - h0.z;
  const float x_prediction = h0.x + tx * dz;
  const float dx = fabs(x_prediction - h2.x);
  const bool tolx_condition = dx < PARAM_TOLERANCE;

  const float y_prediction = h0.y + ty * dz;
  const float dy = fabs(y_prediction - h2.y);
  const bool toly_condition = dy < PARAM_TOLERANCE;

  // Scatter - Updated to last PrPixel
  const float scatterNum = (dx * dx) + (dy * dy);
  const float scatterDenom = 1.f / (h2.z - h1_z);
  const float scatter = scatterNum * scatterDenom * scatterDenom;

  const bool scatter_condition = scatter < MAX_SCATTER;
  const bool condition = tolx_condition && toly_condition && scatter_condition;

  return condition * scatter + !condition * MAX_FLOAT;
}

/**
 * @brief Track following algorithm, loosely based on Pr/PrPixel.
 * @details It should be simplistic in its design, as is the Pixel VELO problem ;)
 *          Triplets are chosen based on a fit and forwarded using a typical track following algo.
 *          Ghosts are inherently out of the equation, as the algorithm considers all possible
 *          triplets and keeps the best. Upon following, if a hit is not found in the adjacent
 *          module, the track[let] is considered complete.
 *          Clones are removed based off a used-hit mechanism. A global array keeps track of
 *          used hits when forming tracks consist of 4 or more hits.
 *
 *          The algorithm consists in two stages: Track following, and seeding. In each step [iteration],
 *          the track following is performed first, hits are marked as used, and then the seeding is performed,
 *          requiring the first two hits in the triplet to be unused.
 * 
 * @param dev_tracks              
 * @param dev_input               
 * @param dev_tracks_to_follow_q1 
 * @param dev_tracks_to_follow_q2 
 * @param dev_hit_used            
 * @param dev_atomicsStorage      
 * @param dev_tracklets           
 * @param dev_weak_tracks         
 * @param dev_event_offsets       
 * @param dev_hit_offsets         
 */
__global__ void searchByTriplet(Track* const dev_tracks, const char* const dev_input,
  int* const dev_tracks_to_follow_q1, int* const dev_tracks_to_follow_q2,
  bool* const dev_hit_used, int* const dev_atomicsStorage, Track* const dev_tracklets,
  int* const dev_weak_tracks, int* const dev_event_offsets, int* const dev_hit_offsets) {
  
  /* Data initialization */
  // Each event is treated with two blocks, one for each side.
  const int event_number = blockIdx.x;
  const int sensor_side = blockIdx.y;
  const int events_under_process = gridDim.x;

  const int tracks_offset = event_number * MAX_TRACKS;
  const int tracks_sides_offset = 2 * event_number * MAX_TRACKS + sensor_side * MAX_TRACKS;

  // Pointers to data within the event
  const int data_offset = dev_event_offsets[event_number];
  const int* const no_sensors = (const int*) &dev_input[data_offset];
  const int* const no_hits = (const int*) (no_sensors + 1);
  const int* const sensor_Zs = (const int*) (no_hits + 1);
  const int number_of_sensors = no_sensors[0];
  const int number_of_hits = no_hits[0];
  const int* const sensor_hitStarts = (const int*) (sensor_Zs + number_of_sensors);
  const int* const sensor_hitNums = (const int*) (sensor_hitStarts + number_of_sensors);
  const unsigned int* const hit_IDs = (const unsigned int*) (sensor_hitNums + number_of_sensors);
  const float* const hit_Xs = (const float*) (hit_IDs + number_of_hits);
  const float* const hit_Ys = (const float*) (hit_Xs + number_of_hits);
  const float* const hit_Zs = (const float*) (hit_Ys + number_of_hits);

  // Per event datatypes
  Track* tracks = &dev_tracks[tracks_offset];
  unsigned int* const tracks_insertPointer = (unsigned int*) dev_atomicsStorage + event_number;

  // Per side datatypes
  const int hit_offset = dev_hit_offsets[event_number];
  bool* const hit_used = dev_hit_used + hit_offset;

  int* const tracks_to_follow_q1 = dev_tracks_to_follow_q1 + tracks_sides_offset;
  int* const tracks_to_follow_q2 = dev_tracks_to_follow_q2 + tracks_sides_offset;
  int* const weak_tracks = dev_weak_tracks + tracks_sides_offset;
  Track* const tracklets = dev_tracklets + tracks_sides_offset;

  // Initialize variables according to event number and sensor side
  // Insert pointers (atomics)
  const int insertPointer_num = 6;
  const int ip_shift = events_under_process + event_number * insertPointer_num * 2 + insertPointer_num * sensor_side;
  // TODO: Maybe convert to dev_atomicsStorage + ip_shift + 1
  unsigned int* const weaktracks_insertPointer = (unsigned int*) dev_atomicsStorage + ip_shift + 1;
  unsigned int* const tracklets_insertPointer = (unsigned int*) dev_atomicsStorage + ip_shift + 2;
  unsigned int* ttf_insertPointer = (unsigned int*) dev_atomicsStorage + ip_shift + 3;
  unsigned int* next_ttf_insertPointer = (unsigned int*) dev_atomicsStorage + ip_shift + 4;
  unsigned int* nh0_to_process = (unsigned int*) dev_atomicsStorage + ip_shift + 5;
  unsigned int* nh1_to_process = (unsigned int*) dev_atomicsStorage + ip_shift + 6;
  unsigned int* temp_ttf_insertPointer; // Just a temp variable to make the exchange

  // Initialize the ttf_insertPointer
  if (threadIdx.x == 0)
    ttf_insertPointer[0] = 0;

  /* The fun begins */
  Track t;
  Sensor s0, s1, s2;
  Hit h0, h1, h2;
  int best_hit_h1, best_hit_h2;

  // extern __shared__ float sh_hits [];
  // float* sh_hit_x = sh_hits;
  // float* sh_hit_y = sh_hit_x + blockDim.x;
  // float* sh_hit_z = sh_hit_y + blockDim.x;
  
  __shared__ float sh_hit_x [64];
  __shared__ float sh_hit_y [64];
  __shared__ float sh_hit_z [64];

  // TODO: Easy to get better, we only need bools
  __shared__ unsigned int sh_h0_process [100];
  __shared__ unsigned int sh_h1_process [100];

  int* tracks_to_follow      = tracks_to_follow_q1;
  int* prev_tracks_to_follow = tracks_to_follow_q2;
  int* temp_tracks_to_follow;

  // Deal with odd or even separately
  int first_sensor = 51 - sensor_side;

  // Prepare s1 and s2 for the first iteration
  const int second_sensor = first_sensor - 2;

  s1.hitStart = sensor_hitStarts[first_sensor];
  s1.hitNums = sensor_hitNums[first_sensor];
  s2.hitStart = sensor_hitStarts[second_sensor];
  s2.hitNums = sensor_hitNums[second_sensor];

  while (first_sensor >= 4) {
    // Iterate in sensors
    // Reuse the info from last sensors
    s0 = s1;
    s1 = s2;

    const int third_sensor = first_sensor - 4;
    s2.hitStart = sensor_hitStarts[third_sensor];
    s2.hitNums = sensor_hitNums[third_sensor];

    // Exchange track_to_follow s
    temp_tracks_to_follow = prev_tracks_to_follow;
    prev_tracks_to_follow = tracks_to_follow;
    tracks_to_follow = temp_tracks_to_follow;

    // Reset the ttf_insertPointer and synchronize
    if (threadIdx.x == 0)
      next_ttf_insertPointer[0] = 0;

    // This syncthreads is required
    __syncthreads();

    // Rotate the ttf pointers
    // Use the same mechanism as in the tracks_to_follow
    temp_ttf_insertPointer = ttf_insertPointer;
    ttf_insertPointer = next_ttf_insertPointer;
    next_ttf_insertPointer = temp_ttf_insertPointer;
    const unsigned int last_ttf_insertPointer = temp_ttf_insertPointer[0];

    // 2a. Track following
    for (int i=0; i<((int) ceilf( ((float) last_ttf_insertPointer) / blockDim.x)); ++i) {
      const unsigned int ttf_element = blockDim.x * i + threadIdx.x;

      // These variables need to go here, shared memory and scope requirements
      float tx, ty;
      int trackno;
      bool track_flag;

      // The logic is broken in two parts for shared memory loading
      const bool ttf_condition = ttf_element < last_ttf_insertPointer;
      if (ttf_condition) {
        const int fulltrackno = prev_tracks_to_follow[ttf_element];
        track_flag = (fulltrackno & 0x80000000) == 0x80000000;
        trackno = fulltrackno & 0x7FFFFFFF;

        const Track* track_pointer = track_flag ? tracklets : tracks;
        t = track_pointer[trackno];

        // Load last two hits in h0, h1
        const int t_hitsNum = t.hitsNum;
        const int h0_num = t.hits[t_hitsNum - 2];
        const int h1_num = t.hits[t_hitsNum - 1];

        h0.x = hit_Xs[h0_num];
        h0.y = hit_Ys[h0_num];
        h0.z = hit_Zs[h0_num];

        h1.x = hit_Xs[h1_num];
        h1.y = hit_Ys[h1_num];
        h1.z = hit_Zs[h1_num];

        // Track following over t, for all hits in the next module
        // Line calculations
        const float td = 1.0f / (h1.z - h0.z);
        const float txn = (h1.x - h0.x);
        const float tyn = (h1.y - h0.y);
        tx = txn * td;
        ty = tyn * td;
      }

      // Search for a best fit
      // Load shared elements
      
      // Iterate in the third list of hits
      // Tiled memory access on h2
      float best_fit = MAX_FLOAT;
      for (int k=0; k<((int) ceilf( ((float) s2.hitNums) / blockDim.x)); ++k){
        
        __syncthreads();
        const int sh_hit_no = blockDim.x * k + threadIdx.x;
        if (sh_hit_no < s2.hitNums){
          const int h2_index = s2.hitStart + sh_hit_no;

          // Coalesced memory accesses
          sh_hit_x[threadIdx.x] = hit_Xs[h2_index];
          sh_hit_y[threadIdx.x] = hit_Ys[h2_index];
          sh_hit_z[threadIdx.x] = hit_Zs[h2_index];
        }
        __syncthreads();

        if (ttf_condition){
          const int last_hit_h2 = min(blockDim.x * k + blockDim.x, s2.hitNums);
          for (int kk=blockDim.x * k; kk<last_hit_h2; ++kk){
            
            const int h2_index = s2.hitStart + kk;
            const int sh_h2_index = kk % blockDim.x;
            h2.x = sh_hit_x[sh_h2_index];
            h2.y = sh_hit_y[sh_h2_index];
            h2.z = sh_hit_z[sh_h2_index];

            const float fit = fitHitToTrack(tx, ty, h0, h1.z, h2);
            const bool fit_is_better = fit < best_fit;

            best_fit = fit_is_better * fit + !fit_is_better * best_fit;
            best_hit_h2 = fit_is_better * h2_index + !fit_is_better * best_hit_h2;
          }
        }
      }

      // We have a best fit!
      // Fill in t, ONLY in case the best fit is acceptable
      if (ttf_condition) {
        if (best_fit != MAX_FLOAT) {
          // Reload h2
          h2.x = hit_Xs[best_hit_h2];
          h2.y = hit_Ys[best_hit_h2];
          h2.z = hit_Zs[best_hit_h2];

          // Mark h2 as used
          hit_used[best_hit_h2] = true;
          
          // Update the tracks to follow, we'll have to follow up
          // this track on the next iteration :)
          t.hits[t.hitsNum++] = best_hit_h2;

          // Update the track in the bag
          if (t.hitsNum > 4){
            // If it is a track made out of *strictly* more than four hits,
            // the trackno refers to the tracks location.
            tracks[trackno] = t;
          }
          else {
            // Otherwise, we have to allocate it in the tracks,
            // and update trackno
            trackno = atomicAdd(tracks_insertPointer, 1);
            tracks[trackno] = t;

            // Also mark the first three as used
            hit_used[t.hits[0]] = true;
            hit_used[t.hits[1]] = true;
            hit_used[t.hits[2]] = true;
          }

          // Add the tracks to the bag of tracks to_follow
          const unsigned int ttfP = atomicAdd(ttf_insertPointer, 1);
          tracks_to_follow[ttfP] = trackno;
        }
        // In the "else" case, we couldn't follow up the track,
        // so we won't be track following it anymore.
        else if (track_flag){
          // If there are only three hits in this track,
          // mark it as "doubtful"
          const unsigned int weakP = atomicAdd(weaktracks_insertPointer, 1);
          weak_tracks[weakP] = trackno;
        }
      }
    }

    if (threadIdx.x == 0) {
      nh0_to_process[0] = 0;
      nh1_to_process[0] = 0;
    }

    __syncthreads();

    // Iterate in all hits for current sensor
    // 2a. Seeding - Track creation
    
    // Pre-seeding 
    // Get the hits we are going to iterate onto in sh_hit_process
    for (int i=0; i<((int) ceilf( ((float) s0.hitNums) / blockDim.x)); ++i) {
      const int element = blockDim.x * i + threadIdx.x;
      if (element < s0.hitNums) {
        const int h0_index = s0.hitStart + element;
        const bool is_h0_used = hit_used[h0_index];

        if (!is_h0_used) {
          const unsigned int htp_pointer = atomicAdd(nh0_to_process, 1);
          sh_h0_process[htp_pointer] = h0_index;
        }
      }
    }
    
    for (int i=0; i<((int) ceilf( ((float) s1.hitNums) / blockDim.x)); ++i) {
      const int element = blockDim.x * i + threadIdx.x;
      if (element < s1.hitNums) {
        const int h1_index = s1.hitStart + element;
        const bool is_h1_used = hit_used[h1_index];

        if (!is_h1_used) {
          const unsigned int htp_pointer = atomicAdd(nh1_to_process, 1);
          sh_h1_process[htp_pointer] = h1_index;
        }
      }
    }

    __syncthreads();

    const unsigned int number_h0_to_process = nh0_to_process[0];
    const unsigned int number_h1_to_process = nh1_to_process[0];
    int h0_index;

    for (int i=0; i<((int) ceilf( ((float) number_h0_to_process) / blockDim.x)); ++i) {
      const int h0_element = blockDim.x * i + threadIdx.x;
      float best_fit = MAX_FLOAT;

      // We repeat this here for performance reasons
      if (h0_element < number_h0_to_process){
        h0_index = sh_h0_process[h0_element];
        h0.x = hit_Xs[h0_index];
        h0.y = hit_Ys[h0_index];
        h0.z = hit_Zs[h0_index];
      }

      for (int j=0; j<number_h1_to_process; ++j) {
        float dxmax, dymax;

        const int h1_index = sh_h1_process[j];
        if (h0_element < number_h0_to_process){
          h1.x = hit_Xs[h1_index];
          h1.y = hit_Ys[h1_index];
          h1.z = hit_Zs[h1_index];

          const float h_dist = fabs((float) ( h1.z - h0.z ));
          dxmax = PARAM_MAXXSLOPE * h_dist;
          dymax = PARAM_MAXYSLOPE * h_dist;
        }

        // Iterate in the third list of hits
        // Tiled memory access on h2
        for (int k=0; k<((int) ceilf( ((float) s2.hitNums) / blockDim.x)); ++k){
          
          __syncthreads();
          const int sh_hit_no = blockDim.x * k + threadIdx.x;
          if (sh_hit_no < s2.hitNums){
            const int h2_index = s2.hitStart + sh_hit_no;

            // Coalesced memory accesses
            sh_hit_x[threadIdx.x] = hit_Xs[h2_index];
            sh_hit_y[threadIdx.x] = hit_Ys[h2_index];
            sh_hit_z[threadIdx.x] = hit_Zs[h2_index];
          }
          __syncthreads();

          if (h0_element < number_h0_to_process){

            const int last_hit_h2 = min(blockDim.x * k + blockDim.x, s2.hitNums);
            for (int kk=blockDim.x * k; kk<last_hit_h2; ++kk){
              
              const int h2_index = s2.hitStart + kk;
              const int sh_h2_index = kk % blockDim.x;
              h2.x = sh_hit_x[sh_h2_index];
              h2.y = sh_hit_y[sh_h2_index];
              h2.z = sh_hit_z[sh_h2_index];

              const float fit = fitHits(h0, h1, h2, dxmax, dymax);
              const bool fit_is_better = fit < best_fit;

              best_fit = fit_is_better * fit + !fit_is_better * best_fit;
              best_hit_h1 = fit_is_better * (h1_index) + !fit_is_better * best_hit_h1;
              best_hit_h2 = fit_is_better * (h2_index) + !fit_is_better * best_hit_h2;
            }
          }
        }
      }

      // We have a best fit! - haven't we?
      const bool accept_track = best_fit != MAX_FLOAT;

      if (accept_track) {
        // Reload h1 and h2
        h1.x = hit_Xs[best_hit_h1];
        h1.y = hit_Ys[best_hit_h1];
        h1.z = hit_Zs[best_hit_h1];

        h2.x = hit_Xs[best_hit_h2];
        h2.y = hit_Ys[best_hit_h2];
        h2.z = hit_Zs[best_hit_h2];

        // Fill in track information
        t.hitsNum = 3;
        t.hits[0] = h0_index;
        t.hits[1] = best_hit_h1;
        t.hits[2] = best_hit_h2;

        // Add the track to the bag of tracks
        const unsigned int trackP = atomicAdd(tracklets_insertPointer, 1);
        tracklets[trackP] = t;

        // Add the tracks to the bag of tracks to_follow
        // Note: The first bit flag marks this is a tracklet (hitsNum == 3),
        // and hence it is stored in tracklets
        const unsigned int ttfP = atomicAdd(ttf_insertPointer, 1);
        tracks_to_follow[ttfP] = 0x80000000 | trackP;
      }
    }

    first_sensor -= 2;
  }

  __syncthreads();

  // Process the last bunch of track_to_follows
  const unsigned int last_ttf_insertPointer = ttf_insertPointer[0];
  for (int i=0; i<((int) ceilf( ((float) last_ttf_insertPointer) / blockDim.x)); ++i) {
    const unsigned int ttf_element = blockDim.x * i + threadIdx.x;

    if (ttf_element < last_ttf_insertPointer) {
      const int fulltrackno = tracks_to_follow[ttf_element];
      const bool track_flag = (fulltrackno & 0x80000000) == 0x80000000;
      const int trackno = fulltrackno & 0x7FFFFFFF;

      // Here we are only interested in three-hit tracks,
      // to mark them as "doubtful"
      if (track_flag) {
        const unsigned int weakP = atomicAdd(weaktracks_insertPointer, 1);
        weak_tracks[weakP] = trackno;
      }
    }
  }

  __syncthreads();

  // Compute the three-hit tracks left
  const unsigned int weaktracks_total = weaktracks_insertPointer[0];
  for (int i=0; i<((int) ceilf( ((float) weaktracks_total) / blockDim.x)); ++i) {
    const unsigned int weaktrack_no = blockDim.x * i + threadIdx.x;
    if (weaktrack_no < weaktracks_total){
      // Load the tracks from the tracklets
      t = tracklets[weak_tracks[weaktrack_no]];

      // Store them in the tracks bag iff they
      // are made out of three unused hits
      if (!hit_used[t.hits[0]] &&
          !hit_used[t.hits[1]] &&
          !hit_used[t.hits[2]]){
        const unsigned int trackno = atomicAdd(tracks_insertPointer, 1);
        tracks[trackno] = t;
      }
    }
  }
}
