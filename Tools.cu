#include "hip/hip_runtime.h"
#include "Tools.cuh"

#include <stdexcept>

// TODO: Remove globals in the short future
int*   h_no_sensors;
int*   h_no_hits;
int*   h_sensor_Zs;
int*   h_sensor_hitStarts;
int*   h_sensor_hitNums;
unsigned int* h_hit_IDs;
float* h_hit_Xs;
float* h_hit_Ys;
float* h_hit_Zs;

void preorder_by_x(std::vector<const std::vector<uint8_t>* > & input) {
  // Order *all* the input vectors by h_hit_Xs natural order
  // per sensor
  const int eventsToProcess = input.size();
  const std::vector<uint8_t>* startingEvent_input = input[0];
  setHPointersFromInput((uint8_t*) &(*startingEvent_input)[0], startingEvent_input->size());

  int number_of_sensors = *h_no_sensors;
  for (int i=0; i<eventsToProcess; ++i) {
    int acc_hitnums = 0;
    const std::vector<uint8_t>* event_input = input[i];
    setHPointersFromInput((uint8_t*) &(*event_input)[0], event_input->size());

    for (int j=0; j<number_of_sensors; j++) {
      const int hitnums = h_sensor_hitNums[j];
      quicksort(h_hit_Xs, h_hit_Ys, h_hit_Zs, h_hit_IDs, acc_hitnums, acc_hitnums + hitnums - 1);
      acc_hitnums += hitnums;
    }
  }
}

void setHPointersFromInput(uint8_t * input, size_t size){
  uint8_t * end = input + size;

  h_no_sensors       = (int32_t*)input; input += sizeof(int32_t);
  h_no_hits          = (int32_t*)input; input += sizeof(int32_t);
  h_sensor_Zs        = (int32_t*)input; input += sizeof(int32_t) * *h_no_sensors;
  h_sensor_hitStarts = (int32_t*)input; input += sizeof(int32_t) * *h_no_sensors;
  h_sensor_hitNums   = (int32_t*)input; input += sizeof(int32_t) * *h_no_sensors;
  h_hit_IDs          = (uint32_t*)input; input += sizeof(uint32_t) * *h_no_hits;
  h_hit_Xs           = (float*)  input; input += sizeof(float)   * *h_no_hits;
  h_hit_Ys           = (float*)  input; input += sizeof(float)   * *h_no_hits;
  h_hit_Zs           = (float*)  input; input += sizeof(float)   * *h_no_hits;

  // if (input != end)
  //   throw std::runtime_error("failed to deserialize event");
}

/**
 * Combine all solutions, in the form:
 *
 * int numberOfEvents
 * per event:
 *     int size
 *
 * per event:
 *     char* output
 *     
 * @param solutions 
 * @param output    
 */
void mergeSolutions(const std::vector<std::vector<char> >& solutions, std::vector<char>& output){
    int numberOfEvents = solutions.size();
    output.resize((numberOfEvents + 1) * sizeof(int));
    char* outputPointer = ((char*) &(output[0]));

    memcpy(outputPointer, &numberOfEvents, sizeof(int));
    outputPointer += sizeof(int);

    int return_size = 0;
    for (int i=0; i<solutions.size(); ++i){
        // Requires lvalue
        int solutions_size = solutions[i].size();
        return_size += solutions_size;
        memcpy(outputPointer, &solutions_size, sizeof(int));
    }
    
    // After resizing, the pointer may change, so recalculate
    output.resize(output.size() + return_size);
    outputPointer = ((char*) &(output[0])) + (numberOfEvents + 1) * sizeof(int);

    for (int i=0; i<solutions.size(); ++i){
        memcpy(outputPointer, &(solutions[i][0]), solutions[i].size());
        outputPointer += solutions[i].size();
    }
}

std::map<std::string, float> calcResults(std::vector<float>& times){
    // sqrt ( E( (X - m)2) )
    std::map<std::string, float> results;
    float deviation = 0.0f, variance = 0.0f, mean = 0.0f, min = float_max(), max = 0.0f;

    for(auto it = times.begin(); it != times.end(); it++){
        const float seconds = (*it);
        mean += seconds;
        variance += seconds * seconds;

        if (seconds < min) min = seconds;
        if (seconds > max) max = seconds;
    }

    mean /= times.size();
    variance = (variance / times.size()) - (mean * mean);
    deviation = std::sqrt(variance);

    results["variance"] = variance;
    results["deviation"] = deviation;
    results["mean"] = mean;
    results["min"] = min;
    results["max"] = max;

    return results;
}

float float_max() {
    const int value = 0x7f800000;
    const float* const fvalue = (const float*) &value;
    return *(float*)& fvalue[0];
}

void quicksort (float* a, float* b, float* c, unsigned int* d, int start, int end) {
    if (start < end) {
        const int pivot = divide(a, b, c, d, start, end);
        quicksort(a, b, c, d, start, pivot - 1);
        quicksort(a, b, c, d, pivot + 1, end);
    }
}

int divide (float* a, float* b, float* c, unsigned int* d, int start, int end) {
    int left;
    int right;
    float pivot;
 
    pivot = a[start];
    left = start;
    right = end;
 
    while (left < right) {
        while (a[right] > pivot) {
            right--;
        }
 
        while ((left < right) && (a[left] <= pivot)) {
            left++;
        }
 
        if (left < right) {
            swap(a[left], a[right]);
            swap(b[left], b[right]);
            swap(c[left], c[right]);
            swap(d[left], d[right]);
        }
    }
 
    swap(a[right], a[start]);
    swap(b[right], b[start]);
    swap(c[right], c[start]);
    swap(d[right], d[start]);
 
    return right;
}

template<typename T>
void swap (T& a, T& b) {
    T temp = a;
    a = b;
    b = temp;
}
