#include "hip/hip_runtime.h"
#include "Tools.cuh"

#include <stdexcept>

// TODO: Remove globals in the short future
int*   h_no_sensors;
int*   h_no_hits;
int*   h_sensor_Zs;
int*   h_sensor_hitStarts;
int*   h_sensor_hitNums;
unsigned int* h_hit_IDs;
float* h_hit_Xs;
float* h_hit_Ys;
float* h_hit_Zs;

void setHPointersFromInput(uint8_t * input, size_t size){
  uint8_t * end = input + size;

  h_no_sensors       = (int32_t*)input; input += sizeof(int32_t);
  h_no_hits          = (int32_t*)input; input += sizeof(int32_t);
  h_sensor_Zs        = (int32_t*)input; input += sizeof(int32_t) * *h_no_sensors;
  h_sensor_hitStarts = (int32_t*)input; input += sizeof(int32_t) * *h_no_sensors;
  h_sensor_hitNums   = (int32_t*)input; input += sizeof(int32_t) * *h_no_sensors;
  h_hit_IDs          = (uint32_t*)input; input += sizeof(uint32_t) * *h_no_hits;
  h_hit_Xs           = (float*)  input; input += sizeof(float)   * *h_no_hits;
  h_hit_Ys           = (float*)  input; input += sizeof(float)   * *h_no_hits;
  h_hit_Zs           = (float*)  input; input += sizeof(float)   * *h_no_hits;

  if (input != end)
    throw std::runtime_error("failed to deserialize event");
}

/**
 * Combine all solutions, in the form:
 *
 * int numberOfEvents
 * per event:
 *     int size
 *
 * per event:
 *     char* output
 *     
 * @param solutions 
 * @param output    
 */
void mergeSolutions(const std::vector<std::vector<char> >& solutions, std::vector<char>& output){
    int numberOfEvents = solutions.size();
    output.resize((numberOfEvents + 1) * sizeof(int));
    char* outputPointer = ((char*) &(output[0]));

    memcpy(outputPointer, &numberOfEvents, sizeof(int));
    outputPointer += sizeof(int);

    int return_size = 0;
    for (int i=0; i<solutions.size(); ++i){
        // Requires lvalue
        int solutions_size = solutions[i].size();
        return_size += solutions_size;
        memcpy(outputPointer, &solutions_size, sizeof(int));
    }
    
    // After resizing, the pointer may change, so recalculate
    output.resize(output.size() + return_size);
    outputPointer = ((char*) &(output[0])) + (numberOfEvents + 1) * sizeof(int);

    for (int i=0; i<solutions.size(); ++i){
        memcpy(outputPointer, &(solutions[i][0]), solutions[i].size());
        outputPointer += solutions[i].size();
    }
}

std::map<std::string, float> calcResults(std::vector<float> times){
    // sqrt ( E( (X - m)2) )
    std::map<std::string, float> results;
    float deviation = 0.0f, variance = 0.0f, mean = 0.0f, min = float_max(), max = 0.0f;

    int n = 0;
    float seconds;
    for(std::vector<float>::iterator it = times.begin(); it != times.end(); it++){
        n++;
        seconds = (*it);
        mean = (mean * (n - 1) + seconds) / n;
        variance += seconds * seconds;

        if (seconds < min) min = seconds;
        if (seconds > max) max = seconds;
    }

    variance = (variance / times.size()) - (mean * mean);
    deviation = std::sqrt(variance);

    results["variance"] = variance;
    results["deviation"] = deviation;
    results["mean"] = mean;
    results["min"] = min;
    results["max"] = max;

    return results;
}

float float_max() {
    const int value = 0x7f800000;
    const float* const fvalue = (const float*) &value;
    return *(float*)& fvalue[0];
}


void quicksort (float* a, float* b, float* c, unsigned int* d, int start, int end) {
    if (start < end) {
        const int pivot = divide(a, b, c, d, start, end);
        quicksort(a, b, c, d, start, pivot - 1);
        quicksort(a, b, c, d, pivot + 1, end);
    }
}

int divide (float* a, float* b, float* c, unsigned int* d, int start, int end) {
    int left;
    int right;
    float pivot;
 
    pivot = a[start];
    left = start;
    right = end;
 
    while (left < right) {
        while (a[right] > pivot) {
            right--;
        }
 
        while ((left < right) && (a[left] <= pivot)) {
            left++;
        }
 
        if (left < right) {
            swap(a[left], a[right]);
            swap(b[left], b[right]);
            swap(c[left], c[right]);
            swap(d[left], d[right]);
        }
    }
 
    swap(a[right], a[start]);
    swap(b[right], b[start]);
    swap(c[right], c[start]);
    swap(d[right], d[start]);
 
    return right;
}

template<typename T>
void swap (T& a, T& b) {
    T temp = a;
    a = b;
    b = temp;
}

